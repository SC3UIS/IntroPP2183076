
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void sumParallel(int *dev_sum, int num)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (tid <= num)
    {

	    // Se calcula el identificador único del hilo sumando el índice del hilo en el bloque con el índice del bloque en la rejilla multiplicado por el tamaño del bloque.
        // Este identificador único se utiliza para determinar qué datos procesa cada hilo.
        // Cada hilo procesará una parte de los datos de entrada.
        // Cada hilo calculará la suma parcial de sus datos asignados.
        // Dependiendo del tamaño del bloque y la rejilla, varios hilos pueden procesar datos diferentes en paralelo.
        // Esta técnica se utiliza para realizar operaciones de suma en paralelo y mejorar el rendimiento en GPUs.
        atomicAdd(dev_sum, (int)tid);

        // Se actualiza el identificador único del hilo sumándole el stride.
        // Esto permite que cada hilo pase al siguiente conjunto de datos que debe procesar.
        tid += stride;
    }
}

int cuda_suma_simple(int num){
    int sum = 0; // Variable para almacenar la suma

    int *dev_sum; // Puntero a la variable de suma en el dispositivo (GPU)

    // Asignar memoria en el dispositivo (GPU) para la variable de suma
    hipMalloc((void**)&dev_sum, sizeof(int));

    // Copiar el valor inicial de la suma desde el host (CPU) al dispositivo (GPU)
    hipMemcpy(dev_sum, &sum, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256; // Número de hilos en un bloque
    int gridSize = (num + blockSize - 1) / blockSize; // Número de bloques en la cuadrícula

    // Ejecutar el kernel sumParallel para realizar la suma en paralelo en el GPU
    sumParallel<<<gridSize, blockSize>>>(dev_sum, num);

    // Copiar el resultado de la suma desde el dispositivo (GPU) al host (CPU)
    hipMemcpy(&sum, dev_sum, sizeof(int), hipMemcpyDeviceToHost);

    // Liberar la memoria asignada en el dispositivo (GPU)
    hipFree(dev_sum);

    return sum; // Devolver la suma calculada
}

int sec_suma_simple(int num){
    int resultado = 0; // Variable para almacenar la suma

    // Ciclo de suma secuencial
    for(int i = 0; i <= num; i++){
         resultado += i; // Sumar el valor de i al resultado
    }

    return resultado; // Devolver la suma calculada
}

float run_cuda_suma_simple(int num){

    hipEvent_t start, stop; // Variables para medir el tiempo de ejecución
    float elapsedTime; // Variable para almacenar el tiempo transcurrido
 
    hipEventCreate(&start); // Crear el evento de inicio
    hipEventCreate(&stop); // Crear el evento de fin
    hipEventRecord(start, 0); // Registrar el inicio de la medición del tiempo

    int result = cuda_suma_simple(num); // Llamar a la función para calcular la suma en GPU
    printf("Resultado Cuda = %d\n", result); // Imprimir el resultado de la suma calculada en GPU

    hipEventRecord(stop, 0); // Registrar el fin de la medición del tiempo
    hipEventSynchronize(stop); // Sincronizar los eventos para asegurar la finalización
    hipEventElapsedTime(&elapsedTime, start, stop); // Calcular el tiempo transcurrido

    hipEventDestroy(start); // Destruir el evento de inicio
    hipEventDestroy(stop); // Destruir el evento de fin
    return elapsedTime; // Devolver el tiempo transcurrido
}

float run_sec_suma_simple(int num){
 
    hipEvent_t start, stop; // Variables para medir el tiempo de ejecución
    float elapsedTime; // Variable para almacenar el tiempo transcurrido

    hipEventCreate(&start); // Crear el evento de inicio
    hipEventCreate(&stop); // Crear el evento de fin
    hipEventRecord(start, 0); // Registrar el inicio de la medición del tiempo

    int result = sec_suma_simple(num); // Llamar a la función para calcular la suma de forma secuencial
    printf("Resultado Secu = %d\n", result); // Imprimir el resultado de la suma calculada de forma secuencial

    hipEventRecord(stop, 0); // Registrar el fin de la medición del tiempo
    hipEventSynchronize(stop); // Sincronizar los eventos para asegurar la finalización
    hipEventElapsedTime(&elapsedTime, start, stop); // Calcular el tiempo transcurrido

    hipEventDestroy(start); // Destruir el evento de inicio
    hipEventDestroy(stop); // Destruir el evento de fin
    return elapsedTime; // Devolver el tiempo transcurrido
}

int main() {
    float run_time_sec, run_time_cuda;
    int num = 0;

    for (int i = 0; i <= 10; i++) {
        printf("\n\n");
	    printf("Comparacion para n = %d\n", num);	

        num = pow(10, i);

        // Ejecutar cálculo de la suma utilizando CUDA
        run_time_cuda = run_cuda_suma_simple(num);

        // Ejecutar cálculo de la suma de forma secuencial
        run_time_sec = run_sec_suma_simple(num);

        printf("Tiempo Cuda (milisegundos)       = %f\n", run_time_cuda); 
        printf("Tiempo Secuencial (milisegundos) = %f\n", run_time_sec); 

        int blockSize = 256;
        int gridSize = (num + blockSize - 1) / blockSize;
        int numThreads = gridSize * blockSize;

        // Calcular el speedup y la escalabilidad
        double speedup = run_time_sec / run_time_cuda;
        double scalability = run_time_sec / (run_time_cuda * numThreads);

        printf("Speedup: %.2f\n", speedup);
        printf("Escalabilidad: %.2f\n", scalability);

        int numOperations = num - 1;  // Número de operaciones de punto flotante realizadas
        double performance = numOperations / (run_time_cuda * 1e6);  // Rendimiento computacional en FLOPS

        printf("Performance: %.2f FLOPS\n", performance);
    }
}

